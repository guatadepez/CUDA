#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define BLOCKSIZE 256
#define WARPSIZE 32

int N=1024;

typedef struct {
    float num;
} Dato;

__inline__ __device__
float warpAllReduceCompare(float val) {
    for (unsigned int mask = WARPSIZE/2; mask > 0; mask /= 2){
        val = fmax(val,__shfl_down(val, mask, WARPSIZE));
    }
    return val;
}

__inline__ __device__
float blockReduceCompare(float val) {
    static __shared__ float shared[WARPSIZE]; // Shared mem for 32 partial sums
    int lane = threadIdx.x % WARPSIZE;
    int wid = threadIdx.x / WARPSIZE;

    val = warpAllReduceCompare(val);     // Each warp performs partial reduction

    if (lane==0) shared[wid]=val; // Write reduced value to shared memory
        __syncthreads();              // Wait for all partial reductions

    //read from shared memory only if that warp existed
    val = (threadIdx.x < blockDim.x / WARPSIZE) ? shared[lane] : 0;

    if (wid==0) val = warpAllReduceCompare(val); //Final reduce within first warp
        return val;
}


__device__ float atomicMaxf(float* address, float val) {
    int *address_as_int =(int*)address;
    int old = *address_as_int, assumed;
    while (val > __int_as_float(old)) {
        assumed = old;
        old = atomicCAS(address_as_int, assumed,
        __float_as_int(val));
    }
    return __int_as_float(old);
}

__global__
void eliteKernel(Dato * device_datos){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float MAX = -1.0f;
    MAX = MAX < device_datos[i].num ? device_datos[i].num : MAX;

    MAX = blockReduceCompare(MAX);
    if(threadIdx.x==0){
        atomicMaxf(&device_datos[0].num, MAX);
    }
}

void fill(Dato *host_datos);
float max(Dato * host_datos);

int main(int argc, char ** argv){
    srand(time(NULL));
    StopWatchInterface *cpu_timer;
    sdkCreateTimer(&cpu_timer);
    sdkResetTimer(&cpu_timer);

    StopWatchInterface *gpu_timer;
    sdkCreateTimer(&gpu_timer);
    sdkResetTimer(&gpu_timer);

    StopWatchInterface *timer;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);

    if(argc==3)
        N = atoi(argv[1]);
    Dato *host_datos;
    Dato *device_datos;

    host_datos = (Dato *) malloc (sizeof(Dato)*N);
    hipMalloc((void**)&device_datos, sizeof(Dato)*N);

    sdkStartTimer(&timer);
    fill(host_datos);
    sdkStopTimer(&timer);
    printf("fill time: %f\n", sdkGetTimerValue(&timer)/1000.0f);

    float FINALMAX = -1;
    if(atoi(argv[2])==1){
        sdkStartTimer(&cpu_timer);
        FINALMAX = max(host_datos);
        sdkStopTimer(&cpu_timer);
        printf("max time: %f\n", sdkGetTimerValue(&cpu_timer)/1000.0f);
    }else{
        int GRIDSIZE = (N+BLOCKSIZE-1)/BLOCKSIZE;
        dim3 block(BLOCKSIZE, 1, 1);
        dim3 grid(GRIDSIZE, 1, 1);

        hipMemcpy(device_datos, host_datos, sizeof(Dato)*N, hipMemcpyHostToDevice);
        sdkStartTimer(&gpu_timer);
        eliteKernel<<<grid,block>>>(device_datos);
        hipDeviceSynchronize();
        hipMemcpy(&FINALMAX, device_datos, sizeof(Dato), hipMemcpyDeviceToHost);
        sdkStopTimer(&gpu_timer);

        printf("max time: %f\n", sdkGetTimerValue(&gpu_timer)/1000.0f);
    }
    printf("MAX = %f\n",FINALMAX);

    return 0;
}

void fill(Dato *host_datos){
    int i;
    for(i=0;i<N;i++){
        host_datos[i].num=(float)(rand()/(RAND_MAX/(99.0-0.1)));
    }
}

float max(Dato * host_datos){
    int i;
    float max = -1;
    for (i=0;i<N;i++){
        if(host_datos[i].num>max)
            max=host_datos[i].num;
    }
    return max;
}
