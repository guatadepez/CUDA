#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define POBLACION 1024
#define LONG_COD 32
#define LIMITE -5.12
#define CROSS_PROBABILITY 0.3
#define MUTATION_PROBABILITY 0.001
#define INTERVALO 10.24/__powf(2,LONG_COD/2)
#define H_INTERVALO 10.24/pow((float)2,(float)LONG_COD/2)

#define BLOCKSIZE 256
#define WARPSIZE 32

int N=1024;

typedef struct {
    char genotipo[LONG_COD];
    float aptitud;
} Individuo;

__host__ __device__ void decoder(float * x, float * y, char * genotipo) {
    int i;
    *x = *y = 0.0;

    #ifdef __CUDA__ARCH__
        // calculo del primer decimal
        for(i=0; i<LONG_COD/2; i++){
            *x += (int)(genotipo[i]) * __powf(2, (LONG_COD/2)-(i+1));
        }
        *x = (*x) * INTERVALO + LIMITE;

        //calculo del segundo decimal
        for(;i<LONG_COD;i++){
            *y += (int)(genotipo[i]) * __powf(2, LONG_COD-(i+1));
        }
        *y = (*y) * INTERVALO + LIMITE;
    #else
        // calculo del primer decimal
        for(i=0; i<LONG_COD/2; i++){
            *x += (int)(genotipo[i]) * pow((float)2, (float)(LONG_COD/2)-(i+1));
        }
        *x = (*x) * H_INTERVALO + LIMITE;

        //calculo del segundo decimal
        for(;i<LONG_COD;i++){
            *y += (int)(genotipo[i]) * pow((float)2, (float)LONG_COD-(i+1));
        }
        *y = (*y) * H_INTERVALO + LIMITE;
    #endif
}

__host__ __device__ float fitness (float p1, float p2){
    return (p1*p1) + (p2*p2);
}

__global__
void tournamentSelectionKernel(Individuo * dev_poblacion, Individuo * dev_selection, hiprandState *dev_state){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<POBLACION){
        hiprandState lstate = dev_state[idx];

        Individuo candidato_a, candidato_b;

        candidato_a = dev_poblacion[(int) (hiprand_uniform(&lstate)*(POBLACION-0.00001))];
        candidato_b = dev_poblacion[(int) (hiprand_uniform(&lstate)*(POBLACION-0.00001))];

        if (candidato_a.aptitud < candidato_b.aptitud)
            dev_selection[idx] = candidato_a;
        else
            dev_selection[idx] = candidato_b;

        dev_state[idx] = lstate;
    }
}

__device__
void sonMutation(Individuo *sons, hiprandState *dev_state, int idx){
    int i,j;
    double randProbability;
    hiprandState lstate = dev_state[idx];
    for(i=0;i<2;i++)
        for(j=0;j<LONG_COD;j++)
            randProbability = (((double) LONG_COD)*hiprand_uniform(&lstate)*(POBLACION-0.00001));
            if(randProbability<MUTATION_PROBABILITY){
                if(sons[i].genotipo[j])
                    sons[i].genotipo[j]=0;
                else
                    sons[i].genotipo[j]=1;
            }
    dev_state[idx] = lstate;
}

__global__
void crossSelectionKernel(Individuo * dev_poblacion, Individuo * dev_selection, hiprandState *dev_state){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<POBLACION-1){
        if(idx==0 || !idx%2){
            hiprandState lstate = dev_state[idx];
            double crossProbability = (((double) LONG_COD)*hiprand_uniform(&lstate)*(POBLACION-0.00001));
            if(crossProbability < MUTATION_PROBABILITY){
                int point, j, aux;
                float x, y;
                point = (int) (((double) LONG_COD)*hiprand_uniform(&lstate)*(POBLACION-0.00001));
                for(j=point; j<LONG_COD; j++){
                    aux=dev_selection[idx].genotipo[j];
                    dev_selection[idx].genotipo[j]=dev_selection[idx+1].genotipo[j];
                    dev_selection[idx+1].genotipo[j]=aux;
                }

                sonMutation(&dev_selection[idx], dev_state, idx);

                decoder(&x, &y, dev_selection[idx].genotipo);
                dev_selection[idx].aptitud=fitness(x,y);

                decoder(&x, &y, dev_selection[idx+1].genotipo);
                dev_selection[idx+1].aptitud=fitness(x,y);
            }
            dev_state[idx] = lstate;
        }
    }
}

__global__
void init_rand(hiprandState *dev_state, int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < POBLACION)
        hiprand_init(idx + seed, 0, 0, &dev_state[idx]);
}

__global__
void init_poblacion(Individuo * dev_poblacion, hiprandState *dev_state){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < POBLACION){
        int i;
        float x, y;
        hiprandState lstate = dev_state[idx];
        for(i=0; i<LONG_COD; i++)
            dev_poblacion[idx].genotipo[i] = hiprand_uniform(&lstate) > 0.5 ? 1.0 : 0.0;
        decoder(&x, &y, dev_poblacion[idx].genotipo);
        dev_poblacion[idx].aptitud = fitness(x,y);
        dev_state[idx] = lstate;
    }
}

/*****************************/
/*****************************/
/*****************************/

__inline__ __device__
float warpAllReduceCompare(float val) {
    for (unsigned int mask = WARPSIZE/2; mask > 0; mask /= 2){
        val = fmin(val,__shfl_down(val, mask, WARPSIZE));
    }
    return val;
}

__inline__ __device__
float blockReduceCompare(float val) {
    static __shared__ float shared[WARPSIZE]; // Shared mem for 32 partial sums
    int lane = threadIdx.x % WARPSIZE;
    int wid = threadIdx.x / WARPSIZE;

    val = warpAllReduceCompare(val);     // Each warp performs partial reduction

    if (lane==0) shared[wid]=val; // Write reduced value to shared memory
        __syncthreads();              // Wait for all partial reductions

    //read from shared memory only if that warp existed
    val = (threadIdx.x < blockDim.x / WARPSIZE) ? shared[lane] : val;

    if (wid==0) val = warpAllReduceCompare(val); //Final reduce within first warp
        return val;
}


__device__ float atomicMaxf(float* address, float val) {
    int *address_as_int =(int*)address;
    int old = *address_as_int, assumed;
    while (val > __int_as_float(old)) {
        assumed = old;
        old = atomicCAS(address_as_int, assumed,
        __float_as_int(val));
    }
    return __int_as_float(old);
}

__device__ float atomicMinf(float* addr, float value) {
    float old = *addr, assumed;

    if(old <= value) return old;

    do {
        assumed = old;
       old = atomicCAS((unsigned int*)addr, __float_as_int(assumed), __float_as_int(value));
    } while(old!=assumed);
    return old;
}

__global__
void eliteKernel(Individuo * dev_seleccion){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float BEST = 1000000.0f;
    BEST = BEST > dev_seleccion[i].aptitud ? dev_seleccion[i].aptitud : BEST;
    BEST = blockReduceCompare(BEST);

    if(threadIdx.x==0){
        atomicMinf(&dev_seleccion[0].aptitud, BEST);
    }
}

/*****************************/
/*****************************/
/*****************************/

void print_selection(Individuo *host_seleccion);
void h_decoder(float * x, float * y, char * genotipo);

int main (int argc, char ** argv) {
    srand(time(NULL));
    printf("[HOST] Starting script\n");

    StopWatchInterface *gpu_timer;
    sdkCreateTimer(&gpu_timer);
    sdkResetTimer(&gpu_timer);

    StopWatchInterface *timer;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);

    if(argc==2)
        N = atoi(argv[1]);

    int GRIDSIZE = (N+BLOCKSIZE-1)/BLOCKSIZE;
    dim3 block(BLOCKSIZE, 1, 1);
    dim3 grid(GRIDSIZE, 1, 1);

    Individuo BEST;
    unsigned int generation = 0;

    /*
    * Random initialization.
    **/
    hiprandState *dev_state;
    hipMalloc(&dev_state, sizeof(hiprandState)*POBLACION);
    init_rand<<<grid, block>>>(dev_state, rand());

    Individuo * host_seleccion, * host_poblacion;
    Individuo * dev_seleccion, * dev_poblacion;

    host_poblacion = (Individuo *) malloc (sizeof(Individuo)*POBLACION);
    host_seleccion = (Individuo *) malloc (sizeof(Individuo)*POBLACION);
    hipMalloc((void**)&dev_poblacion, sizeof(Individuo)*POBLACION);
    hipMalloc((void**)&dev_seleccion, sizeof(Individuo)*POBLACION);

    sdkStartTimer(&timer);
    init_poblacion<<<grid, block>>>(dev_poblacion, dev_state);
    sdkStopTimer(&timer);
    printf("fill time: %f\n", sdkGetTimerValue(&timer)/1000.0f);
    //hipMemcpy(host_seleccion, dev_poblacion, sizeof(Individuo)*POBLACION, hipMemcpyDeviceToHost);
    //print_selection(host_seleccion);

    sdkStartTimer(&gpu_timer);
    do{
        tournamentSelectionKernel<<<grid, block>>>(dev_poblacion, dev_seleccion, dev_state);
        crossSelectionKernel<<<grid, block>>>(dev_poblacion, dev_seleccion, dev_state);
        eliteKernel<<<grid,block>>>(dev_seleccion);
        hipMemcpy(host_seleccion, dev_seleccion, sizeof(Individuo)*POBLACION, hipMemcpyDeviceToHost);
        //print_selection(host_seleccion);
        //getchar();
        generation++;

        hipDeviceSynchronize();
        hipMemcpy(&BEST, dev_seleccion, sizeof(Individuo), hipMemcpyDeviceToHost);
        //printf("\nbest aptitud: %f\n", BEST.aptitud);
    }while(BEST.aptitud > pow(10,-2));

    eliteKernel<<<grid,block>>>(dev_seleccion);
    hipMemcpy(&BEST, dev_seleccion, sizeof(Individuo), hipMemcpyDeviceToHost);
    float x, y;
    h_decoder(&x, &y, BEST.genotipo);

    sdkStopTimer(&gpu_timer);
    printf("max time: %f\n", sdkGetTimerValue(&gpu_timer)/1000.0f);

    printf ("*************************************\n");
    printf ("*          FIN DEL ALGORITMO        *\n");
    printf ("*************************************\n");
    printf (" - En el punto (%.5f, %.5f)\n", x, y);
    printf (" - Su fenotipo es %.5f\n", BEST.aptitud);
    printf (" - Es la generacion numero %i\n", generation);
    printf ("*************************************\n");

    free(host_poblacion);
    free(host_seleccion);
    hipFree(dev_poblacion);
    hipFree(dev_seleccion);
    hipFree(dev_state);

    hipDeviceReset();
    return 0;
}

void print_selection(Individuo *host_seleccion){
    int i;
    for(i=0; i<POBLACION; i++){
        printf("\nhost_seleccion[%d] = %f", i, host_seleccion[i].aptitud);
    }
}

void h_decoder(float * x, float * y, char * genotipo) {
    int i;
    *x = *y = 0.0;

    // calculo del primer decimal
    for(i=0; i<LONG_COD/2; i++){
        *x += (int)(genotipo[i]) * pow((float)2, (float)(LONG_COD/2)-(i+1));
    }
    *x = (*x) * H_INTERVALO + LIMITE;

    //calculo del segundo decimal
    for(;i<LONG_COD;i++){
        *y += (int)(genotipo[i]) * pow((float)2, (float)LONG_COD-(i+1));
    }
    *y = (*y) * H_INTERVALO + LIMITE;
}
